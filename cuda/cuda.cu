#include <iostream>
#include <vector>
#include <queue>
#include <limits>
#include <cmath>
#include <ctime>
#include <algorithm>
#include <bitset>
#include <string>
#include "hip/hip_runtime.h"


using namespace std;

const int INF = numeric_limits<int>::max();
const int THREADS_PER_BLOCK = 256;

vector<vector<int>> generate_matrix(int n) {
    int rate = (int)(0.5 * (n * (n - 1) / 2));
    vector<vector<int>> matrix(n, vector<int>(n, INF));

    for (int i = 0; i < n; ++i) {
        matrix[i][i] = 0;
    }

    while (rate > 0) {
        int v1 = rand() % n;
        int v2 = rand() % n;
        if (v1 != v2 && matrix[v1][v2] == INF) {
            int value = 1 + rand() % 30;
            matrix[v1][v2] = value;
            matrix[v2][v1] = value;
            --rate;
        }
    }

    return matrix;
}

__device__ void dijkstra(int* matrix, int* dist, bool* visited, int n, int start) {
    for (int i = 0; i < n; i++) {
        dist[i] = INF;
        visited[i] = false;
    }
    dist[start] = 0;

    for (int count = 0; count < n - 1; count++) {
        int u = -1;
        for (int i = 0; i < n; i++) {
            if (!visited[i] && (u == -1 || dist[i] < dist[u])) {
                u = i;
            }
        }

        visited[u] = true;
        for (int v = 0; v < n; v++) {
            if (matrix[u * n + v] != INF && matrix[u * n + v] != 0 && dist[u] + matrix[u * n + v] < dist[v]) {
                dist[v] = dist[u] + matrix[u * n + v];
            }
        }
    }
}

__device__ void find_shortest_paths(int* matrix, int* shortest_paths, int n) {
    for (int start = 0; start < n; start++) {
        int* dist = new int[n];
        bool* visited = new bool[n];
        dijkstra(matrix, dist, visited, n, start);
        for (int i = 0; i < n; i++) {
            shortest_paths[start * n + i] = dist[i];
        }
        delete[] dist;
        delete[] visited;
    }
}

__device__ bool DFS_check_device(int* matrix, int n) {
    bool* visited = new bool[n];
    for (int i = 0; i < n; i++) {
        visited[i] = false;
    }

    auto DFS = [&](int v, auto& DFS_ref) -> void {
        visited[v] = true;
        for (int i = 0; i < n; ++i) {
            if (matrix[v * n + i] != INF && matrix[v * n + i] != 0 && !visited[i]) {
                DFS_ref(i, DFS_ref);
            }
        }
    };

    DFS(0, DFS);

    bool all_visited = true;
    for (int i = 0; i < n; i++) {
        if (!visited[i]) {
            all_visited = false;
            break;
        }
    }

    delete[] visited;
    return all_visited;
}

__device__ bool check_the_limit_device(int* paths, int* new_paths, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            if (new_paths[i * n + j] > 1.5 * paths[i * n + j]) {
                return false;
            }
        }
    }
    return true;
}

__global__ void count_permutations_kernel(int* d_matrix, int* d_shortest_paths, int* d_result, int n, int edges_count, int* permutations, int* d_optimal_matrix) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= (1 << edges_count)) return;

    int* h_new_matrix = new int[n * n];
    for (int i = 0; i < n * n; i++) {
        h_new_matrix[i] = d_matrix[i];
    }

    int edge = 0;
    for (int i = 0; i < n; i++) {
        for (int j = i + 1; j < n; j++) {
            if (h_new_matrix[i * n + j] != INF && h_new_matrix[i * n + j] != 0) {
                if (!(permutations[idx * edges_count + edge])) {
                    h_new_matrix[i * n + j] = INF;
                    h_new_matrix[j * n + i] = INF;
                }
                edge++;
            }
        }
    }

    if (DFS_check_device(h_new_matrix, n)) {
        int* d_new_paths = new int[n * n];
        find_shortest_paths(h_new_matrix, d_new_paths, n);

        if (check_the_limit_device(d_shortest_paths, d_new_paths, n)) {
            int edges_removed = 0;
            for (int j = 0; j < edges_count; j++) {
                if (!permutations[idx * edges_count + j]) {
                    edges_removed++;
                }
            }
            if (edges_removed > atomicMax(d_result, edges_removed)) {
                for (int i = 0; i < n * n; i++) {
                    d_optimal_matrix[i] = h_new_matrix[i];
                }
            }
        }

        delete[] d_new_paths;
    }

    delete[] h_new_matrix;
}

pair<int, vector<vector<int>>> delete_edges_brute_force(const vector<vector<int>>& matrix) {
    int n = matrix.size();
    int edges_count = 0;
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            if (matrix[i][j] != INF && matrix[i][j] != 0) {
                edges_count++;
            }
        }
    }
    edges_count /= 2;

    int* h_matrix;
    hipMallocManaged(&h_matrix, n * n * sizeof(int));
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            h_matrix[i * n + j] = matrix[i][j];
        }
    }

    int* d_shortest_paths;
    hipMallocManaged(&d_shortest_paths, n * n * sizeof(int));

    int* h_shortest_paths = new int[n * n];
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            h_shortest_paths[i * n + j] = INF;
        }
    }

    for (int start = 0; start < n; start++) {
        vector<int> dist(n, INF);
        vector<bool> visited(n, false);
        dist[start] = 0;

        for (int count = 0; count < n - 1; count++) {
            int u = -1;
            for (int i = 0; i < n; i++) {
                if (!visited[i] && (u == -1 || dist[i] < dist[u])) {
                    u = i;
                }
            }

            visited[u] = true;
            for (int v = 0; v < n; v++) {
                if (matrix[u][v] != INF && matrix[u][v] != 0 && dist[u] + matrix[u][v] < dist[v]) {
                    dist[v] = dist[u] + matrix[u][v];
                }
            }
        }

        for (int i = 0; i < n; i++) {
            h_shortest_paths[start * n + i] = dist[i];
        }
    }

    hipMemcpy(d_shortest_paths, h_shortest_paths, n * n * sizeof(int), hipMemcpyHostToDevice);

    int max_edges_removed = 0;

    int total_permutations = 1 << edges_count;
    int* permutations;
    hipMallocManaged(&permutations, total_permutations * edges_count * sizeof(int));
    for (int i = 0; i < total_permutations; ++i) {
        for (int j = 0; j < edges_count; ++j) {
            permutations[i * edges_count + j] = (i >> j) & 1;
        }
    }

    int* d_result;
    hipMallocManaged(&d_result, sizeof(int));
    hipMemcpy(d_result, &max_edges_removed, sizeof(int), hipMemcpyHostToDevice);

    int* d_optimal_matrix;
    hipMallocManaged(&d_optimal_matrix, n * n * sizeof(int));

    clock_t start_time = clock();
    count_permutations_kernel<<<(total_permutations + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(
        h_matrix, d_shortest_paths, d_result, n, edges_count, permutations, d_optimal_matrix);
    hipDeviceSynchronize();
    clock_t stop_time = clock();

    hipMemcpy(&max_edges_removed, d_result, sizeof(int), hipMemcpyDeviceToHost);

    cout << "Parallel execution time: " << (double)(stop_time - start_time) / CLOCKS_PER_SEC << " seconds." << endl;

    vector<vector<int>> optimal_matrix(n, vector<int>(n));
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            optimal_matrix[i][j] = d_optimal_matrix[i * n + j];
        }
    }

    hipFree(permutations);
    hipFree(h_matrix);
    hipFree(d_shortest_paths);
    hipFree(d_result);
    hipFree(d_optimal_matrix);
    delete[] h_shortest_paths;

    return {max_edges_removed, optimal_matrix};
}

int main() {
    srand(time(nullptr));

    int n = 10;
    cout << "Number of vertices: " << n << "\n";
    vector<vector<int>> matrix = generate_matrix(n);

    cout << "Initial matrix:\n";
    for (const auto& row : matrix) {
        for (int val : row) {
            cout << (val == INF ? "INF" : to_string(val)) << " ";
        }
        cout << "\n";
    }

    clock_t start_time = clock();
    pair<int, vector<vector<int>>> result = delete_edges_brute_force(matrix);
    int bruteforce_solution = result.first;
    vector<vector<int>> optimal_matrix = result.second;
    clock_t stop_time = clock();

    cout << "\nOptimal number of edges removed: " << bruteforce_solution << "\n";
    if (bruteforce_solution > 0) {
        cout << "Optimal matrix:\n";
        for (const auto& row : optimal_matrix) {
            for (int val : row) {
                cout << (val == INF ? "INF" : to_string(val)) << " ";
            }
            cout << "\n";
        }
    } else {
        cout << "No edges could be removed while maintaining constraints.\n";
    }

    double elapsed_time = (double)(stop_time - start_time) / CLOCKS_PER_SEC;
    cout << "Brute force execution time: " << elapsed_time << " seconds\n";

    return 0;
}
